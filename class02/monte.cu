#include<stdio.h>
#include<time.h>
#include<stdlib.h>
#include<hiprand/hiprand_kernel.h>
#include<hip/hip_runtime.h>


#define TRIALS_PER_THREAD 4096
#define BLOCKS 256
#define THREADS 256
#define PI 3.1415926535989


#define CUDA_CALL(x) do { if(x!= hipSuccess) {\
   printf("Error at %s:%d -- %s \n", __FILE__,__LINE__, hipGetErrorString(x)); \
   return EXIT_FAILURE;}} while(0)


__global__ void monteWithGPU( hiprandState *states, float * estimate) {
	double x, y;
    int toss, number_in_circle=0;
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int seed = id;
    hiprand_init(1234, seed ,0 ,&states[id]);

	for(toss=0; toss <TRIALS_PER_THREAD; toss++) {
        /*curand_uniform() range 0~1*/
		x = hiprand_uniform(&states[id])*2 - 1;
        y = hiprand_uniform(&states[id])*2 - 1;
	    
        if (x*x+y*y <= 1.0f)
            number_in_circle++;
	}	
    estimate[id] = 4.0f *number_in_circle / (TRIALS_PER_THREAD) ;
}

int main(void) {
	double pi_estimate;
	//time variables
	clock_t start, end;
	double cpu_time_used;
    float number_in_circle = 0;
    hiprandState *devStates;
    float *dev, *host;

	start = clock();

    host = (float*)malloc(sizeof(float)* THREADS * BLOCKS );
    hipMalloc((void**) &dev, BLOCKS * THREADS * sizeof(float));
    hipMalloc((void**)&devStates, THREADS * BLOCKS * sizeof(hiprandState));
    monteWithGPU<<<BLOCKS, THREADS>>>(devStates, dev);
    hipMemcpy(host, dev, BLOCKS * THREADS * sizeof(float), hipMemcpyDeviceToHost);

    for (int i=0; i < BLOCKS * THREADS ; i++){
        number_in_circle+=host[i];
    }

    pi_estimate = number_in_circle/ (BLOCKS * THREADS);
	end = clock();
	cpu_time_used = ((double)(end - start)) /CLOCKS_PER_SEC;
	printf("pi_estimate = %f \n", pi_estimate);
	printf("Elapsed time = %f seconds \n", cpu_time_used);
    return 0;	
}
