
#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <hiprand/hiprand_kernel.h>
#define CURAND_CALL(x) do { \
    if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \ 
    return EXIT_FAILURE;}} while(0)

int main(int argc, char *argv[]) {
    int n = 100; 
    int i; 
    hiprandGenerator_t gen; 
    float *devData, *hostData; 
    
    /* Allocate n floats on host */ 
    hostData = (float *)calloc(n, sizeof(float)); 
    
    /* Allocate n floats on device */ 
    hipMalloc(&devData, n*sizeof(float));

    CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    
    /* Set seed */ 
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 1234567));
    
    /* Generate n floats on device */ 
    CURAND_CALL(hiprandGenerateUniform(gen, devData, n));
    /* Copy device memory to host */ 
    hipMemcpy(hostData, devData, n * sizeof(float), hipMemcpyDeviceToHost);
    printf("hello world\n");    
    /* Show result */ 
    for(i = 0; i < n; i++) { 
        printf("%1.4f ", hostData[i]); }
    printf("\n"); 
    /* Cleanup */ 
    CURAND_CALL(hiprandDestroyGenerator(gen)); 
    hipFree(devData); 
    free(hostData); 
    return 0;
}
